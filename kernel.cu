#include "hip/hip_runtime.h"
// Expression Template Operators.cpp : Defines the entry point for the console application.

#include "hip/hip_runtime.h"
#include <iostream>

#include <Stopwatch.h>
#include <RuntimeTest.h>
#include <Algebra\Vector_Operators.h>

#include <thrust\device_vector.h>
#include "DeviceOperators.h"

#include "iterators.h"
#include "vector.h"
#include "operators.h"
#include "expressions.h"

thrust::device_vector<double> etree_thrust_result;
thrust::device_vector<double> thrust_std_result;
std::vector<double> etree_loop_result;
std::vector<double> etree_stl_result;
std::vector<double> std_result;
std::vector<double> loop_result;

const double pi = 3.14159265358979323846;
const double pi2 = pi*pi;

using std::endl;

#define DURATION(timer) timer.nanoseconds().to_seconds()

template <typename V, typename Out>
double test(const V* _x, const V* _y, Out* output)
{
	using namespace vector_operators::vector;
	using namespace thrust_operators::vector;
	using namespace etree::operators::binary;
	using namespace etree::operators::unary;

	Stopwatch timer;

	const std::size_t N = _x->size();
	const V& x = *_x;
	const V& y = *_y;

	timer.start();
	const V result = cos(y)*x + y - x;
	timer.stop();

	*output = result;
	return DURATION(timer);
}

template <typename V, typename Out>
double test_Loop(const V* _x, const V* _y, Out* output)
{
	using T = typename V::value_type;

	Stopwatch timer;

	const std::size_t N = _x->size();
	const V& x = *_x;
	const V& y = *_y;

	timer.start();
	T* result = new T[N];
	for (unsigned int j = 0; j < N; j++)
		result[j] = y[j] * x[j] + y[j] - x[j];

	timer.stop();

	output->resize(N);
	std::copy(result, result + N, output->begin());

	delete[] result;
	return DURATION(timer);
}

template <typename V>
void setup(const std::size_t N, V* x, V* y)
{
	using T = typename V::value_type;

	x->resize(N);
	y->resize(N);

	x->assign(N, -316376.1);
	y->assign(N, 362178);
}

void runTests()
{
	typedef double T;

	typedef std::vector<T> Vec;
	typedef thrust::device_vector<T> DVec;

	std::vector<std::size_t> sizes = {
		static_cast<std::size_t>(1e1),
		static_cast<std::size_t>(1e2),
		static_cast<std::size_t>(1e3),
		static_cast<std::size_t>(1e4),
		static_cast<std::size_t>(1e5),
		static_cast<std::size_t>(2e5),
		static_cast<std::size_t>(3e5),
		static_cast<std::size_t>(4e5),
		static_cast<std::size_t>(5e5),
		static_cast<std::size_t>(6e5),
		static_cast<std::size_t>(7e5),
		static_cast<std::size_t>(1e6),
		static_cast<std::size_t>(2e6),
		static_cast<std::size_t>(3e6),/*
		static_cast<std::size_t>(4e6),
		static_cast<std::size_t>(5e6),
		static_cast<std::size_t>(1e7),
		static_cast<std::size_t>(2e7),
		static_cast<std::size_t>(3e7),*/
	};

	Vec x, y;
	DVec d_x, d_y;
	etree::vector<T> c_x, c_y;
	etree::vector<T, etree::constructors::STL> stl_x, stl_y;
	etree::vector<T, etree::constructors::Thrust, DVec, etree::parallel_policy> ed_x, ed_y;

	RuntimeTest<double> etree_loop(sizes), etree_stl(sizes), etree_thrust(sizes), thrust_std(sizes), std(sizes);

	etree_loop.storeSetup(setup<decltype(c_x)>, &c_x, &c_y);
	etree_loop.storeTest("Operators_ETree_Loop_times.txt", test<decltype(c_x), Vec>, &c_x, &c_y, &etree_loop_result);
	etree_loop.runAll(10);
	etree_loop.save();

	//etree_stl.storeSetup(setup<decltype(stl_x)>, &stl_x, &stl_y);
	//etree_stl.storeTest("Operators_ETree_STL_times.txt", test<decltype(stl_x), Vec>, &stl_x, &stl_y, &etree_stl_result);
	//etree_stl.runAll(10);
	//etree_stl.save();

	thrust_std.storeSetup(setup<decltype(d_x)>, &d_x, &d_y);
	thrust_std.storeTest("Operators_Thrust_Std_times.txt", test<decltype(d_x), DVec>, &d_x, &d_y, &thrust_std_result);
	thrust_std.runAll(10);
	thrust_std.save();

	etree_thrust.storeSetup(setup<decltype(ed_x)>, &ed_x, &ed_y);
	etree_thrust.storeTest("Operators_ETree_Thrust_times.txt", test<decltype(ed_x), DVec>, &ed_x, &ed_y, &etree_thrust_result);
	etree_thrust.runAll(10);
	etree_thrust.save();

	std.storeSetup(setup<Vec>, &x, &y);
	std.storeTest("Operators_STD_times.txt", test<Vec, Vec>, &x, &y, &std_result);
	std.storeTest("Operators_Loop_times.txt", test_Loop<Vec, Vec>, &x, &y, &loop_result);
	std.runAll(10);
	std.save();

	Vec etree_thrust_host(etree_thrust_result.size());
	Vec thrust_std_host(etree_thrust_result.size());

	thrust::copy(etree_thrust_result.begin(), etree_thrust_result.end(), etree_thrust_host.begin());
	thrust::copy(thrust_std_result.begin(), thrust_std_result.end(), thrust_std_host.begin());

	std::cout << "Equality of Answers:"
		<< endl
		<< "loop_result == etree_loop_result: "
		<< std::to_string(loop_result == etree_loop_result)
		<< endl

		/*<< "loop_result == etree_stl_result: "
		<< std::to_string(loop_result == etree_stl_result)
		<< endl*/

		<< "loop_result == etree_thrust_result: "
		<< std::to_string(loop_result == thrust_std_host)
		<< endl

		<< "loop_result == thrust_std_result: "
		<< std::to_string(loop_result == etree_thrust_host)
		<< endl

		<< "loop_result == std_result: "
		<< std::to_string(loop_result == std_result)
		<< endl;

	std::cout << "STD: "		<< std_result[900]		  << endl
			  << "Loop: "		<< loop_result[900]		  << endl
			  << "Thrust STD: " << thrust_std_host[900] << endl
			  << "ETree Thrust: " << etree_thrust_host[900] << endl
			  //<< "ETree STL: "	<< etree_stl_result[900]  << endl
			  << "ETree Loop: " << etree_loop_result[900] << endl;

	std::cout << "Result vector sizes:"		<< endl;
	std::cout << "loop_result size: "		<< loop_result.size() << endl;
	std::cout << "std_result size: "		<< std_result.size() << endl;
	std::cout << "etree_loop_result size: " << etree_loop_result.size() << endl;
	std::cout << "thrust_std_host size: " << thrust_std_host.size() << endl;
	std::cout << "etree_thrust_result size: " << etree_thrust_host.size() << endl;
	//std::cout << "etree_stl_result size: "	<< etree_stl_result.size() << endl;
}

int main()
{
	using namespace etree::operators::binary;
	//using namespace etree::operators::unary;

	hipFree(0);
	using T		= double;
	using Vec	= std::vector<T>;
	using DVec	= thrust::device_vector<T>;
	using EVec  = etree::vector < T, etree::constructors::STL, Vec, etree::serial_policy>;
	using EDVec = etree::vector<T, etree::constructors::Thrust, DVec, etree::parallel_policy>;

	const std::size_t N = 3;

	EDVec dx(N), dy(N);
	dx.assign(N, -1);
	dy.assign(N, 3);


	EDVec dsum = dx + dy;

	/*EVec x(N), y(N);
	x.assign(N, -1);
	y.assign(N, 3);

	EVec sum = x + y;*/

	//DVec& data = sum;

	//std::cout << "Sum = " << host[0] << std::endl;
	//std::cout << "Sum = " << sum[0] << std::endl;
	runTests();
	return 0;
}


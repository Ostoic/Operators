#include "hip/hip_runtime.h"
// Expression Template Operators.cpp : Defines the entry point for the console application.

#include "hip/hip_runtime.h"
#include <iostream>

#include <Stopwatch.h>
#include <RuntimeTest.h>
#include <Algebra\Vector_Operators.h>

#include <thrust\device_vector.h>
#include "DeviceOperators.h"

#include "iterators.h"
#include "vector.h"
#include "operators.h"
#include "expressions.h"

thrust::device_vector<double> vap_thrust_result;
thrust::device_vector<double> thrust_std_result;
std::vector<double> vap_loop_result;
std::vector<double> vap_stl_result;
std::vector<double> std_result;
std::vector<double> loop_result;

const double pi = 3.14159265358979323846;
const double pi2 = pi*pi;

using std::endl;

#define DURATION(timer) timer.nanoseconds().to_seconds()

template <typename V, typename Out>
double test(const V* _x, const V* _y, Out* output)
{
	using namespace vector_operators::vector;
	using namespace thrust_operators::vector;
	using namespace vap::operators::binary;
	using namespace vap::operators::unary;

	Stopwatch timer;

	const std::size_t N = _x->size();
	const V& x = *_x;
	const V& y = *_y;

	timer.start();
	const V result = cos(y)*x + y - x;
	timer.stop();

	*output = result;
	return DURATION(timer);
}

template <typename V, typename Out>
double test_Loop(const V* _x, const V* _y, Out* output)
{
	using T = typename V::value_type;

	Stopwatch timer;

	const std::size_t N = _x->size();
	const V& x = *_x;
	const V& y = *_y;

	timer.start();
	T* result = new T[N];
	for (unsigned int j = 0; j < N; j++)
		result[j] = y[j] * x[j] + y[j] - x[j];

	timer.stop();

	output->resize(N);
	std::copy(result, result + N, output->begin());

	delete[] result;
	return DURATION(timer);
}

template <typename V>
void setup(const std::size_t N, V* x, V* y)
{
	using T = typename V::value_type;

	x->resize(N);
	y->resize(N);

	x->assign(N, -316376.1);
	y->assign(N, 362178);
}

void runTests()
{
	typedef double T;

	typedef std::vector<T> Vec;
	typedef thrust::device_vector<T> DVec;

	std::vector<std::size_t> sizes = {
		static_cast<std::size_t>(1e1),
		static_cast<std::size_t>(1e2),
		static_cast<std::size_t>(1e3),
		static_cast<std::size_t>(1e4),
		static_cast<std::size_t>(1e5),
		static_cast<std::size_t>(2e5),
		static_cast<std::size_t>(3e5),
		static_cast<std::size_t>(4e5),
		static_cast<std::size_t>(5e5),
		static_cast<std::size_t>(6e5),
		static_cast<std::size_t>(7e5),
		static_cast<std::size_t>(1e6),
		static_cast<std::size_t>(2e6),
		static_cast<std::size_t>(3e6),/*
		static_cast<std::size_t>(4e6),
		static_cast<std::size_t>(5e6),
		static_cast<std::size_t>(1e7),
		static_cast<std::size_t>(2e7),
		static_cast<std::size_t>(3e7),*/
	};

	Vec x, y;
	DVec d_x, d_y;
	vap::vector<T> c_x, c_y;
	vap::vector<T, vap::constructors::STL> stl_x, stl_y;
	vap::vector<T, vap::constructors::Thrust, DVec, vap::parallel_policy> ed_x, ed_y;

	RuntimeTest<double> vap_loop(sizes), vap_stl(sizes), vap_thrust(sizes), thrust_std(sizes), std(sizes);

	vap_loop.storeSetup(setup<decltype(c_x)>, &c_x, &c_y);
	vap_loop.storeTest("Operators_vap_Loop_times.txt", test<decltype(c_x), Vec>, &c_x, &c_y, &vap_loop_result);
	vap_loop.runAll(10);
	vap_loop.save();

	//vap_stl.storeSetup(setup<decltype(stl_x)>, &stl_x, &stl_y);
	//vap_stl.storeTest("Operators_vap_STL_times.txt", test<decltype(stl_x), Vec>, &stl_x, &stl_y, &vap_stl_result);
	//vap_stl.runAll(10);
	//vap_stl.save();

	thrust_std.storeSetup(setup<decltype(d_x)>, &d_x, &d_y);
	thrust_std.storeTest("Operators_Thrust_Std_times.txt", test<decltype(d_x), DVec>, &d_x, &d_y, &thrust_std_result);
	thrust_std.runAll(10);
	thrust_std.save();

	vap_thrust.storeSetup(setup<decltype(ed_x)>, &ed_x, &ed_y);
	vap_thrust.storeTest("Operators_vap_Thrust_times.txt", test<decltype(ed_x), DVec>, &ed_x, &ed_y, &vap_thrust_result);
	vap_thrust.runAll(10);
	vap_thrust.save();

	std.storeSetup(setup<Vec>, &x, &y);
	std.storeTest("Operators_STD_times.txt", test<Vec, Vec>, &x, &y, &std_result);
	std.storeTest("Operators_Loop_times.txt", test_Loop<Vec, Vec>, &x, &y, &loop_result);
	std.runAll(10);
	std.save();

	Vec vap_thrust_host(vap_thrust_result.size());
	Vec thrust_std_host(vap_thrust_result.size());

	thrust::copy(vap_thrust_result.begin(), vap_thrust_result.end(), vap_thrust_host.begin());
	thrust::copy(thrust_std_result.begin(), thrust_std_result.end(), thrust_std_host.begin());

	std::cout << "Equality of Answers:"
		<< endl
		<< "loop_result == vap_loop_result: "
		<< std::to_string(loop_result == vap_loop_result)
		<< endl

		/*<< "loop_result == vap_stl_result: "
		<< std::to_string(loop_result == vap_stl_result)
		<< endl*/

		<< "loop_result == vap_thrust_result: "
		<< std::to_string(loop_result == thrust_std_host)
		<< endl

		<< "loop_result == thrust_std_result: "
		<< std::to_string(loop_result == vap_thrust_host)
		<< endl

		<< "loop_result == std_result: "
		<< std::to_string(loop_result == std_result)
		<< endl;

	std::cout << "STD: "		<< std_result[900]		  << endl
			  << "Loop: "		<< loop_result[900]		  << endl
			  << "Thrust STD: " << thrust_std_host[900] << endl
			  << "vap Thrust: " << vap_thrust_host[900] << endl
			  //<< "vap STL: "	<< vap_stl_result[900]  << endl
			  << "vap Loop: " << vap_loop_result[900] << endl;

	std::cout << "Result vector sizes:"		<< endl;
	std::cout << "loop_result size: "		<< loop_result.size() << endl;
	std::cout << "std_result size: "		<< std_result.size() << endl;
	std::cout << "vap_loop_result size: " << vap_loop_result.size() << endl;
	std::cout << "thrust_std_host size: " << thrust_std_host.size() << endl;
	std::cout << "vap_thrust_result size: " << vap_thrust_host.size() << endl;
	//std::cout << "vap_stl_result size: "	<< vap_stl_result.size() << endl;
}

int main()
{
	using namespace vap::operators::binary;
	//using namespace vap::operators::unary;

	hipFree(0);
	using T		= double;
	using Vec	= std::vector<T>;
	using DVec	= thrust::device_vector<T>;
	using EVec  = vap::vector < T, vap::constructors::STL, Vec, vap::serial_policy>;
	using EDVec = vap::vector<T, vap::constructors::Thrust, DVec, vap::parallel_policy>;

	const std::size_t N = 3;

	EDVec dx(N), dy(N);
	dx.assign(N, -1);
	dy.assign(N, 3);


	EDVec dsum = dx + dy;

	/*EVec x(N), y(N);
	x.assign(N, -1);
	y.assign(N, 3);

	EVec sum = x + y;*/

	//DVec& data = sum;

	//std::cout << "Sum = " << host[0] << std::endl;
	//std::cout << "Sum = " << sum[0] << std::endl;
	runTests();
	return 0;
}

